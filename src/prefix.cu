#include "hip/hip_runtime.h"
#include "prefix.cuh"

namespace cusr {
    namespace program {

        using namespace std;

        static float constant_prob = 0.2;

        /**
         * Each seed is initialized by the real random generator engine.
         * Update the seed after it has been used a given number of times.
         */
        static int seed_using_times = 2000;

        static int seed_count = seed_using_times;

        void set_seed_using_times(int time) {
            seed_using_times = time;
        }

        void set_constant_prob(float p_const) {
            constant_prob = p_const;
        }

        int gen_rand_int(int loBound, int upBound) {
            if (seed_count-- <= 0) {
                seed_count = seed_using_times;
                std::random_device rd;
                std::mt19937 gen(rd());
                std::uniform_int_distribution<int> dis(loBound, upBound);
                srand(dis(gen));
            }
            int bound_width = upBound - loBound + 1;
            return rand() % bound_width + loBound;
        }

        float gen_rand_float(float loBound, float upBound) {
            if (seed_count-- <= 0) {
                seed_count = seed_using_times;
                std::random_device rd;
                std::mt19937 gen(rd());
                std::uniform_int_distribution<int> dis(loBound, upBound);
                srand(dis(gen));
            }
            float rd = loBound + (float) (rand()) / (float) (RAND_MAX / (upBound - loBound));
            return rd;
        }

        int get_depth_of_prefix(prefix_t &prefix) {
            stack<int> s;
            for (int i = prefix.size() - 1; i >= 0; i--) {
                Node &node = prefix[i];
                if (node.node_type == NodeType::VAR || node.node_type == NodeType::CONST) {
                    s.push(0);
                } else if (node.node_type == NodeType::BFUNC) {
                    int child1 = s.top();
                    s.pop();
                    int child2 = s.top();
                    s.pop();
                    int max_depth = child1 >= child2 ? child1 : child2;
                    s.push(max_depth + 1);
                } else {
                    s.top() += 1;
                }
            }
            return s.top() + 1;
        }

        int rand_roulette_pos(prefix_t &prefix, bool allow_terminal) {
            int len = prefix.size();

            auto *weights = new float[len];
            float total = 0;
            for (int i = 0; i < len; i++) {
                if (prefix[i].node_type == NodeType::BFUNC || prefix[i].node_type == NodeType::UFUNC) {
                    weights[i] = FUNCTION_WEIGHTS;
                    total += FUNCTION_WEIGHTS;
                } else {
                    weights[i] = TERMINAL_WEIGHTS;
                    total += TERMINAL_WEIGHTS;
                }
            }
            for (int i = 0; i < len; i++) {
                weights[i] /= total;
            }
            for (int i = 1; i < len; i++) {
                weights[i] += weights[i - 1];
            }
            float rand_float = gen_rand_float(0, 1);

            int pos;
            while (true) {
                pos = 0;
                for (; pos < len; pos++) {
                    if (rand_float <= weights[pos] || pos == len - 1) {
                        break;
                    }
                }
                if (!allow_terminal && (prefix[pos].node_type == NodeType::VAR || prefix[pos].node_type == NodeType::CONST)) {
                    rand_float = gen_rand_float(0, 1);
                    continue;
                }
                break;
            }
            delete[] weights;
            return pos;
        }

        pair<int, int> rand_subtree_index_roulette(prefix_t &prefix, bool allow_terminal) {
            int pos = rand_roulette_pos(prefix, allow_terminal);

            if (prefix[pos].node_type == NodeType::CONST || prefix[pos].node_type == NodeType::VAR) {
                return {pos, pos + 1};
            }
            int op_count = 0;
            int num_count = 0;
            int end = pos;
            for (; end < prefix.size(); end++) {
                Node &node = prefix[end];
                if (node.node_type == NodeType::BFUNC) {
                    op_count++;
                } else if (node.node_type == NodeType::VAR || node.node_type == NodeType::CONST) {
                    num_count++;
                } else // [ node.node_type == NodeType::UFUNC ]
                {
                    continue;
                }
                if (op_count + 1 == num_count) {
                    break;
                }
            }
            return {pos, end + 1};
        }

        void rand_constant(Node &node, pair<float, float> &range) {
            node.node_type = NodeType::CONST;
            node.constant = gen_rand_float(range.first, range.second);
        }

        void rand_variable(Node &node, int variable_num) {
            node.node_type = NodeType::VAR;
            node.variable = gen_rand_int(0, variable_num - 1);
        }

        void rand_terminal(Node &node, pair<float, float> &range, int variable_num, float p_constant) {
            float rand_float = gen_rand_float(0, 1);
            if (rand_float <= p_constant) {
                rand_constant(node, range);
            } else {
                rand_variable(node, variable_num);
            }
        }

        void rand_terminal(Node &node, pair<float, float> &range, int variable_num) {
            rand_terminal(node, range, variable_num, constant_prob);
        }

        void rand_function(Node &node, vector<Function> &function_set) {
            int len = function_set.size();
            int rand_int = gen_rand_int(0, len - 1);
            func_t rand_func = function_set[rand_int];
            node.function = rand_func;
            if (rand_func == Function::ADD || rand_func == Function::SUB ||
                rand_func == Function::MUL || rand_func == Function::DIV ||
                rand_func == Function::MAX || rand_func == Function::MIN) {
                node.node_type = NodeType::BFUNC;
            } else /** if (rand_func == Function::SIN || rand_func == Function::COS || rand_func == Function::TAN ||
        rand_func == Function::LOG || rand_func == Function::INV) */
            {
                node.node_type = NodeType::UFUNC;
            }
        }

        TreeNode *
        gen_full_init_tree(int depth, pair<float, float> &range, vector<Function> &func_set, int variable_num) {
            if (depth == 1) {
                auto *tree_node = new TreeNode();
                rand_terminal(tree_node->node, range, variable_num);
                return tree_node;
            }
            auto *tree_node = new TreeNode();
            rand_function(tree_node->node, func_set);
            if (tree_node->node.node_type == NodeType::BFUNC) {
                tree_node->left = gen_full_init_tree(depth - 1, range, func_set, variable_num);
                tree_node->right = gen_full_init_tree(depth - 1, range, func_set, variable_num);
            } else {
                tree_node->left = gen_full_init_tree(depth - 1, range, func_set, variable_num);
            }
            return tree_node;
        }

        static bool is_first_rand = true;

#define RETURN_RATE 0.1

        TreeNode *
        gen_growth_init_tree(int depth, pair<float, float> &range, vector<Function> &func_set, int variable_num) {
            if (depth == 1) {
                auto *tree_node = new TreeNode();
                rand_terminal(tree_node->node, range, variable_num);
                return tree_node;
            }

            float rand_float = gen_rand_float(0, 1);

            if (!is_first_rand) {
                if (rand_float <= RETURN_RATE) // if return now
                {
                    auto *tree_node = new TreeNode();
                    rand_terminal(tree_node->node, range, variable_num);
                    return tree_node;
                }
            }

            is_first_rand = false;

            auto *tree_node = new TreeNode();
            rand_function(tree_node->node, func_set);

            if (tree_node->node.node_type == NodeType::BFUNC) {
                tree_node->left = gen_growth_init_tree(depth - 1, range, func_set, variable_num);
                tree_node->right = gen_growth_init_tree(depth - 1, range, func_set, variable_num);
            } else {
                tree_node->left = gen_growth_init_tree(depth - 1, range, func_set, variable_num);
            }

            return tree_node;
        }

        void get_init_prefix(prefix_t &prefix, TreeNode *tree_node) {
            if (tree_node == nullptr) {
                return;
            }
            prefix.emplace_back(tree_node->node);
            get_init_prefix(prefix, tree_node->left);
            delete tree_node->left;
            get_init_prefix(prefix, tree_node->right);
            delete tree_node->right;
        }

        static string function_to_string(Function function) {
            switch (function) {
                case Function::ADD:
                    return "+";

                case Function::SUB:
                    return "-";

                case Function::MUL:
                    return "*";

                case Function::DIV:
                    return "/";

                case Function::MAX:
                    return "max";

                case Function::MIN:
                    return "min";

                case Function::SIN:
                    return "sin";

                case Function::COS:
                    return "cos";

                case Function::TAN:
                    return "tan";

                case Function::LOG:
                    return "log";

                case Function::INV:
                    return "inv";

                default:
                    return "error";
            }
        }

        string prefix_to_infix(prefix_t &prefix) {
            stack<string> s;
            for (int i = prefix.size() - 1; i >= 0; i--) {
                Node &node = prefix[i];
                if (node.node_type == NodeType::CONST) {
                    s.push(std::to_string(node.constant));
                } else if (node.node_type == NodeType::VAR) {
                    string var = "x";
                    var.append(std::to_string(node.variable));
                    s.push(var);
                } else if (node.node_type == NodeType::BFUNC) {
                    string tmp = "(";
                    tmp.append(s.top()).append(" ");
                    s.pop();
                    tmp.append(function_to_string(node.function));
                    tmp.append(" ").append(s.top()).append(")");
                    s.pop();
                    s.push(tmp);
                } else {
                    string tmp;
                    tmp = function_to_string(node.function);
                    if (s.top().size() == 1) {
                        tmp.append(s.top());
                    } else {
                        tmp.append("(").append(s.top()).append(")");
                    }
                    s.pop();
                    s.push(tmp);
                }
            }
            return s.top();
        }

        string prefix_to_string(prefix_t &prefix) {
            string ret;

            for (int i = 0; i < prefix.size(); i++) {
                auto node = prefix[i];
                if (node.node_type == NodeType::UFUNC || node.node_type == NodeType::BFUNC) {
                    ret.append(function_to_string(node.function)).append(" ");
                } else if (node.node_type == NodeType::VAR) {
                    ret.append("x").append(to_string(node.variable)).append(" ");
                } else {
                    ret.append(to_string(node.constant)).append(" ");
                }
            }
            return ret;
        }

        pair<int, int> get_subtree_index(prefix_t &prefix, int start_pos) {
            int len = prefix.size();

            // if the pos is a terminal, it is the subtree
            if (prefix[start_pos].node_type == NodeType::CONST || prefix[start_pos].node_type == NodeType::VAR) {
                return {start_pos, start_pos + 1};
            }

            // if the pos is not a terminal, we find the corresponding subtree
            int op_count = 0;
            int num_count = 0;
            int end = start_pos;

            for (; end < len; end++) {
                Node &node = prefix[end];

                if (node.node_type == NodeType::BFUNC) {
                    op_count++;
                } else if (node.node_type == NodeType::VAR || node.node_type == NodeType::CONST) {
                    num_count++;
                } else // if (node.node_type == NodeType::UFUNC)
                {
                    continue;
                }
                if (op_count + 1 == num_count) {
                    break;
                }
            }
            return {start_pos, end + 1};
        }
    }
}