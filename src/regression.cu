#include "hip/hip_runtime.h"
#include "regression.cuh"

namespace cusr {

    using namespace std;
    using namespace program;
    using namespace fit;

    void RegressionEngine::fit(vector<vector<float>> &dataset, vector<float> &label) {
        this->dataset = dataset;
        this->label = label;
        cusr::program::set_constant_prob(this->p_constant);
        do_fit_init();

        clock_t iter_begin = clock();

        do_population_init();
        update_population_attributes();

        printf("%15s %15s %15s %15s %15s %15s\n",
               "gen", "best fit", "best len", "best dep", "max len", "max dep");
        printf("---------------------------------------------------");
        printf("---------------------------------------------------\n");

        printf("%15d %15.5f %15d %15d %15d %15d\n",
               0, best_program.fitness, best_program.length, best_program.depth, max_length_in_population,
               max_depth_in_population);

        int iter_times = 1;

        while (true) {
            gen_next_generation();
            update_population_attributes();

            printf("%15d %15.5f %15d %15d %15d %15d\n",
                   iter_times, best_program.fitness, best_program.length, best_program.depth, max_length_in_population,
                   max_depth_in_population);

            if (++iter_times >= generations || this->best_program.fitness <= this->stopping_criteria) {
                break;
            }
        }
        this->regress_time_in_sec = (float) (clock() - iter_begin) / (float) CLOCKS_PER_SEC;
        printf("---------------------------------------------------");
        printf("---------------------------------------------------\n");
        cout << "> iteration time: " << regress_time_in_sec << "s" << endl;
        cout << "> best program:   " << prefix_to_infix(best_program.prefix) << endl << endl << endl;

        if (use_gpu) {
            freeDataSetAndLabel(&device_dataset);
        }
    }

    void RegressionEngine::do_fit_init() {
        assert(!dataset.empty() && dataset.size() == label.size());

        this->variable_nums = dataset[0].size();

        if(this->n_hall_of_fame > this->population_size) {
            this->n_hall_of_fame = this->population_size;
            cout << "automatically change n_hall_of_fame = " << this->n_hall_of_fame << endl;
        }

        if(this->n_components > n_hall_of_fame) {
            this->n_components = this->n_hall_of_fame;
            cout << "automatically change n_components = " << this->n_components << endl;
        }

        if (use_gpu) {
            do_gpu_init();
        }
    }

    void RegressionEngine::do_population_init() {
        this->population.clear();

        // full initialize
        if (this->init_method == InitMethod::full) {
            for (int i = 0; i < population_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(*gen_full_init_program(depth, const_range, function_set, variable_nums));
            }
        }

        // growth initialize
        if (this->init_method == InitMethod::growth) {
            for (int i = 0; i < population_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(
                        *gen_growth_init_program(depth, const_range, function_set, variable_nums));
            }
        }

        // ramped half and half
        if (this->init_method == InitMethod::half_and_half) {
            // assert(population_size >= 2);
            int full_size = population_size / 2;
            int growth_size = population_size - full_size;

            for (int i = 0; i < full_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(*gen_full_init_program(depth, const_range, function_set, variable_nums));
            }

            for (int i = 0; i < growth_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(
                        *gen_growth_init_program(depth, const_range, function_set, variable_nums));
            }
        }

        if (use_gpu) {
            calculate_population_fitness_gpu();
        } else {
            calculate_population_fitness_cpu();
        }
    }

    Program RegressionEngine::do_mutation(Program &program) {
        Program ret;

        float rand_float = gen_rand_float(0, 1);

        if (rand_float < p_crossover) {
            int index = tournament_selection_cpu(population, tournament_size, parsimony_coefficient);
            ret = crossover_mutation(program, population[index]);
        } else if (rand_float < p_crossover + p_hoist_mutation) {
            ret = hoist_mutation(program);
        } else if (rand_float < p_crossover + p_hoist_mutation + p_point_mutation) {
            ret = point_mutation(program, function_set, const_range, variable_nums);
        } else if (rand_float < p_crossover + p_hoist_mutation + p_point_mutation + p_subtree_mutation) {
            int rand_int = gen_rand_int(init_depth.first, init_depth.second);
            ret = subtree_mutation(program, rand_int, const_range, function_set, variable_nums);
        } else if (rand_float <
                   p_crossover + p_hoist_mutation + p_point_mutation + p_subtree_mutation + p_point_replace) {
            ret = point_replace_mutation(program, function_set, const_range, variable_nums);
        } else {
            return program;
        }

        ret.depth = get_depth_of_prefix(ret.prefix);

        // hoist until the depth under the specified depth
        while (restrict_depth && ret.depth > max_program_depth) {
            ret = hoist_mutation(ret);
            ret.depth = get_depth_of_prefix(ret.prefix);
        }

        ret.length = ret.prefix.size();
        return ret;
    }

    void RegressionEngine::gen_next_generation() {
        vector<Program> next_gen;

        // elite strategy
        int best_fitness_index = 0;
        for (int i = 1; i < population_size; i++) {
            if (population[i].fitness < population[best_fitness_index].fitness) {
                best_fitness_index = i;
            }
        }

        next_gen.emplace_back(population[best_fitness_index]);

        // selection and do mutation
        for (int i = 1; i < population_size; i++) {
            int index = tournament_selection_cpu(population, tournament_size, parsimony_coefficient);
            next_gen.emplace_back(do_mutation(population[index]));
        }

        population.assign(next_gen.begin(), next_gen.end());

        // fitness evaluation
        if (use_gpu) {
            calculate_population_fitness_gpu();
        } else {
            calculate_population_fitness_cpu();
        }

    }

    void RegressionEngine::update_population_attributes() {

        int best_fitness_index = 0;
        int max_prefix_length = 0;
        int max_prefix_depth = 0;

        for (int i = 1; i < population_size; i++) {
            if (population[i].fitness < population[best_fitness_index].fitness) {
                best_fitness_index = i;
            }
            if (population[i].length > max_prefix_length) {
                max_prefix_length = population[i].length;
            }
            if (population[i].depth > max_prefix_depth) {
                max_prefix_depth = population[i].depth;
            }
        }

        this->best_program = population[best_fitness_index];
        this->max_length_in_population = max_prefix_length;
        this->max_depth_in_population = max_prefix_depth;
        this->best_program_in_each_gen.emplace_back(this->best_program);
    }

    void RegressionEngine::fit(vector<vector<float>> &dataset, vector<float> &label, string corr) {

        fit(dataset, label);

        clock_t time1 = clock();

        // argsort all programs by their fitnesses
        vector<int> indices(population_size);
        iota(indices.begin(), indices.end(), 0);
        partial_sort(indices.begin(), indices.begin()+n_hall_of_fame, indices.end(),
        [this](int i, int j) { return this->population[i].fitness < this->population[j].fitness; });

        clock_t time2 = clock();

        // calculate predictions of hall_of_fame programs(from best to worst)
        // TODO: implement predict_gpu() and check whether use gpu or cpu to calculate
        vector<vector<float>> predictions(n_hall_of_fame, vector<float>(dataset.size()));
        for(int i=0; i<n_hall_of_fame; i++) {
            predict_cpu(&population[indices[i]], dataset, dataset.size(), this->metric, predictions[i]);
        }

        clock_t time3 = clock();

        // calculate correlations between hall_of_fame programs(may add new metrics in the future, design the
        // code structure such that is easy to maintain!)
        // TODO: check metric type 'pearson' or 'spearman'
        vector<vector<float>> corr_matrix(n_hall_of_fame, vector<float>(n_hall_of_fame));
        cal_corr_matrix(predictions, corr_matrix);

        clock_t time4 = clock();
        
        // select top n_components most uncorrelated programs from
        // population[indices[0]] ... population[indices[n_hall_of_fame-1]]
        // by iteratively excluding the worse program (the larger index j) of a most correlated pair of programs
        unordered_set<int> excluded;
        int to_exclude;
        float max_corr;
        while(n_hall_of_fame - excluded.size() > n_components) {
            to_exclude = 0;
            max_corr = 0.0;
            for(int i=0; i<n_hall_of_fame-1; i++) {
                if(excluded.find(i) != excluded.end()) continue;
                for(int j=i+1; j<n_hall_of_fame; j++) {
                    if(excluded.find(j) != excluded.end()) continue;
                    if(corr_matrix[i][j] > max_corr) {
                        max_corr = corr_matrix[i][j];
                        to_exclude = j;
                    }
                    else if(corr_matrix[i][j] == max_corr && j > to_exclude) to_exclude = j; 
                }
            }
            excluded.insert(to_exclude);
        }

        clock_t time5 = clock();

        // save top n_components most uncorrelated programs to components
        for(int i=0; i<n_hall_of_fame; i++) {
            if(excluded.find(i)==excluded.end()) components.emplace_back(population[indices[i]]);
        }

        clock_t time6 = clock();
        cout << "> iteration time1: " << (float) (time2 - time1) / (float) CLOCKS_PER_SEC << "s" << endl;
        cout << "> iteration time2: " << (float) (time3 - time2) / (float) CLOCKS_PER_SEC << "s" << endl;
        cout << "> iteration time3: " << (float) (time4 - time3) / (float) CLOCKS_PER_SEC << "s" << endl;
        cout << "> iteration time4: " << (float) (time5 - time4) / (float) CLOCKS_PER_SEC << "s" << endl;
        cout << "> iteration time5: " << (float) (time6 - time5) / (float) CLOCKS_PER_SEC << "s" << endl;

    }

    void RegressionEngine::transform(vector<vector<float>> &dataset, vector<vector<float>> &new_dataset) {
        // TODO: check dataset.size()==new_dataset.size() and dataset.size() + n_components == new_dataset.size()
    }

    void RegressionEngine::calculate_population_fitness_cpu() {
        for (int i = 0; i < population_size; i++) {
            calculate_fitness_cpu(&population[i], dataset, label, dataset.size(), this->metric);
        }
    }

    void RegressionEngine::calculate_population_fitness_gpu() {
        int blockNum = (dataset.size() - 1) / THREAD_PER_BLOCK + 1;
        calculatePopulationFitness(this->device_dataset, blockNum, population, this->metric);
    }

    void RegressionEngine::do_gpu_init() {
        copyDatasetAndLabel(&device_dataset, dataset, label);
    }

    RegressionEngine::~RegressionEngine() {
        freeDataSetAndLabel(&this->device_dataset);
    }

    void cal_corr_matrix(const vector<vector<float>> &data, vector<vector<float>> &corr_matrix) {
        // TODO: check corr_matrix.size() == corr_matrix[0].size() == data.size()

        int n_data = data.size();
        int data_size = data[0].size();
        
        vector<float> data_mean(n_data);
        for(int i=0; i<n_data; i++) {
            data_mean[i] = accumulate(data[i].begin(), data[i].end(), 0.0) / (data_size - 1);
        }

        vector<vector<float>> data_sum_prod_dev(n_data, vector<float>(n_data));
        for(int k=0; k<data_size; k++) {
            for(int i=0; i<n_data; i++) {
                for(int j=i; j<n_data; j++) {
                    data_sum_prod_dev[i][j] += (data[i][k] - data_mean[i])*(data[j][k] - data_mean[j]);
                }
            }
        }
        for(int i=0; i<n_data; i++) {
            data_sum_prod_dev[i][i] = sqrt(data_sum_prod_dev[i][i] / data_size);
        }

        for(int i=0; i<n_data; i++) {
            for(int j=i+1; j<n_data; j++) {
                if(data_sum_prod_dev[i][j]==0) corr_matrix[i][j] = 0;
                else corr_matrix[i][j] = 
                            data_sum_prod_dev[i][j] / (data_size-1) / data_sum_prod_dev[i][i] / data_sum_prod_dev[j][j];
            }
        }
    }
}